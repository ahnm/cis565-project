#include "hip/hip_runtime.h"
#include "common.h"
#include "commonCUDA.cuh"
#include <hip/hip_math_constants.h>

__device__ static void dspace
	(
	int irez,
	double d2201,  double d2211,  double d3210,   double d3222,  double d4410,
	double d4422,  double d5220,  double d5232,   double d5421,  double d5433,
	double dedt,   double del1,   double del2,    double del3,   double didt,
	double dmdt,   double dnodt,  double domdt,   double argpo,  double argpdot,
	double t,      double tc,     double gsto,    double xfact,  double xlamo,
	double no,
	double& atime, double& em,    double& argpm,  double& inclm, double& xli,
	double& mm,    double& xni,   double& nodem,  double& dndt,  double& nm
	);


__global__ void sgp4(satelliterecord_soa_t *satrec, int N, double tsince, float4 *r)
{
#define STRIDE		0
#define OFFSET		0
#define GROUP_SIZE	512
	int block_start_idx = blockIdx.x * blockDim.x;
	//int tid = block_start_idx + ((threadIdx.x + OFFSET) % STRIDE);
	int tid = block_start_idx + threadIdx.x;
	if(tid < N){
		double	am		,	axnl	,	aynl	,	betal	,	cos2u	,	coseo1	,	cosip	,
				cosisq	,	delm	,	delomg	,	em		,	emsq	,	ecose	,	el2		,
				eo1		,	ep		,	esine	,	argpm	,	argpp	,	argpdf	,	pl		,
				mrt		,	rl		,	sin2u	,	sineo1	,	sinip	,	su		,	t2		,
				t3		,	t4		,	tem5	,	temp	,	temp1	,	temp2	,	tempa	,
				tempe	,	templ	,	u		,	ux		,	uy		,	uz		,	inclm	,
				mm		,	nm		,	nodem	,	xinc	,	xincp	,	xl		,	xlm		,
				mp		,	xmdf	,	xmx		,	xmy		,	nodedf	,	xnode	,	nodep	,
				tc		,	dndt	,	x2o3;
		int ktr;

		/* ------------------ set mathematical constants --------------- */
		// sgp4fix divisor for divide by zero check on inclination
		//const double temp4    =   1.0 + cos(HIP_PI-1.0e-9);
		//twopi = 2.0 * HIP_PI;
		x2o3  = 2.0 / 3.0;
		// sgp4fix identify constants and allow alternate values
		//vkmpersec     = gravity_constants.radiusearthkm * gravity_constants.xke/60.0;
	
		/* --------------------- clear sgp4 error flag ----------------- */
		satrec[tid].t     = tsince;
		satrec[tid].error = 0;

		/* ------- update for secular gravity and atmospheric drag ----- */
		xmdf    = satrec[tid].mo + satrec[tid].mdot * satrec[tid].t;
		argpdf  = satrec[tid].argpo + satrec[tid].argpdot * satrec[tid].t;
		nodedf  = satrec[tid].nodeo + satrec[tid].nodedot * satrec[tid].t;
		argpm   = argpdf;
		mm      = xmdf;
		t2      = satrec[tid].t * satrec[tid].t;
		nodem   = nodedf + satrec[tid].nodecf * t2;
		tempa   = 1.0 - satrec[tid].cc1 * satrec[tid].t;
		tempe   = satrec[tid].bstar * satrec[tid].cc4 * satrec[tid].t;
		templ   = satrec[tid].t2cof * t2;

		if (satrec[tid].isimp != 1)
		{
			delomg = satrec[tid].omgcof * satrec[tid].t;
			delm   = satrec[tid].xmcof *
				(pow((1.0 + satrec[tid].eta * cos(xmdf)), 3) -
				satrec[tid].delmo);
			temp   = delomg + delm;
			mm     = xmdf + temp;
			argpm  = argpdf - temp;
			t3     = t2 * satrec[tid].t;
			t4     = t3 * satrec[tid].t;
			tempa  = tempa - satrec[tid].d2 * t2 - satrec[tid].d3 * t3 -
				satrec[tid].d4 * t4;
			tempe  = tempe + satrec[tid].bstar * satrec[tid].cc5 * (sin(mm) -
				satrec[tid].sinmao);
			templ  = templ + satrec[tid].t3cof * t3 + t4 * (satrec[tid].t4cof +
				satrec[tid].t * satrec[tid].t5cof);
		}

		nm    = satrec[tid].no;
		em    = satrec[tid].ecco;
		inclm = satrec[tid].inclo;
		if (satrec[tid].method == 'd')
		{
			tc = satrec[tid].t;
			dspace
				(
				satrec[tid].irez,
				satrec[tid].d2201, satrec[tid].d2211, satrec[tid].d3210,
				satrec[tid].d3222, satrec[tid].d4410, satrec[tid].d4422,
				satrec[tid].d5220, satrec[tid].d5232, satrec[tid].d5421,
				satrec[tid].d5433, satrec[tid].dedt,  satrec[tid].del1,
				satrec[tid].del2,  satrec[tid].del3,  satrec[tid].didt,
				satrec[tid].dmdt,  satrec[tid].dnodt, satrec[tid].domdt,
				satrec[tid].argpo, satrec[tid].argpdot, satrec[tid].t, tc,
				satrec[tid].gsto, satrec[tid].xfact, satrec[tid].xlamo,
				satrec[tid].no, satrec[tid].atime,
				em, argpm, inclm, satrec[tid].xli, mm, satrec[tid].xni,
				nodem, dndt, nm
				);
		} // if method = d

		if (nm <= 0.0)
		{
			//         printf("# error nm %f\n", nm);
			satrec[tid].error = 2;
		}
		am = pow((gravity_constants.xke / nm),x2o3) * tempa * tempa;
		nm = gravity_constants.xke / pow(am, 1.5);
		em = em - tempe;

		// fix tolerance for error recognition
		if ((em >= 1.0) || (em < -0.001) || (am < 0.95))
		{
			//         printf("# error em %f\n", em);
			satrec[tid].error = 1;
		}
		if (em < 0.0)
			em  = 1.0e-6;
		mm     = mm + satrec[tid].no * templ;
		xlm    = mm + argpm + nodem;
		emsq   = em * em;
		temp   = 1.0 - emsq;

		nodem  = fmod(nodem, 2.0 * HIP_PI);
		argpm  = fmod(argpm, 2.0 * HIP_PI);
		xlm    = fmod(xlm, 2.0 * HIP_PI);
		mm     = fmod(xlm - argpm - nodem, 2.0 * HIP_PI);

		/* ----------------- compute extra mean quantities ------------- */
		/*sinim = sin(inclm);
		cosim = cos(inclm);*/

		/* -------------------- add lunar-solar periodics -------------- */
		ep     = em;
		xincp  = inclm;
		argpp  = argpm;
		nodep  = nodem;
		mp     = mm;
		sinip  = sin(inclm);
		cosip  = cos(inclm);
		if (satrec[tid].method == 'd')
		{
			dpper
				(
				satrec[tid].e3,   satrec[tid].ee2,  satrec[tid].peo,
				satrec[tid].pgho, satrec[tid].pho,  satrec[tid].pinco,
				satrec[tid].plo,  satrec[tid].se2,  satrec[tid].se3,
				satrec[tid].sgh2, satrec[tid].sgh3, satrec[tid].sgh4,
				satrec[tid].sh2,  satrec[tid].sh3,  satrec[tid].si2,
				satrec[tid].si3,  satrec[tid].sl2,  satrec[tid].sl3,
				satrec[tid].sl4,  satrec[tid].t,    satrec[tid].xgh2,
				satrec[tid].xgh3, satrec[tid].xgh4, satrec[tid].xh2,
				satrec[tid].xh3,  satrec[tid].xi2,  satrec[tid].xi3,
				satrec[tid].xl2,  satrec[tid].xl3,  satrec[tid].xl4,
				satrec[tid].zmol, satrec[tid].zmos, satrec[tid].inclo,
				'n', ep, xincp, nodep, argpp, mp
				);
			if (xincp < 0.0)
			{
				xincp  = -xincp;
				nodep = nodep + HIP_PI;
				argpp  = argpp - HIP_PI;
			}
			if ((ep < 0.0 ) || ( ep > 1.0))
			{
				//            printf("# error ep %f\n", ep);
				satrec[tid].error = 3;
			}
		} // if method = d

		/* -------------------- long period periodics ------------------ */
		if (satrec[tid].method == 'd')
		{
			sinip =  sin(xincp);
			cosip =  cos(xincp);
			satrec[tid].aycof = -0.5*gravity_constants.j3oj2*sinip;
			// sgp4fix for divide by zero for xincp = 180 deg
			if (fabs(cosip+1.0) > 1.5e-12)
				satrec[tid].xlcof = -0.25 * gravity_constants.j3oj2 * sinip * (3.0 + 5.0 * cosip) / (1.0 + cosip);
			else
				satrec[tid].xlcof = -0.25 * gravity_constants.j3oj2 * sinip * (3.0 + 5.0 * cosip) / (1.0 + cos(HIP_PI-1.0e-9));
		}
		axnl = ep * cos(argpp);
		temp = 1.0 / (am * (1.0 - ep * ep));
		aynl = ep* sin(argpp) + temp * satrec[tid].aycof;
		xl   = mp + argpp + nodep + temp * satrec[tid].xlcof * axnl;

		/* --------------------- solve kepler's equation --------------- */
		u    = fmod(xl - nodep, 2.0 * HIP_PI);
		eo1  = u;
		tem5 = 9999.9;
		ktr = 1;
		//   sgp4fix for kepler iteration
		//   the following iteration needs better limits on corrections
		while (( fabs(tem5) >= 1.0e-12) && (ktr <= 10) )
		{
			sineo1 = sin(eo1);
			coseo1 = cos(eo1);
			tem5   = 1.0 - coseo1 * axnl - sineo1 * aynl;
			tem5   = (u - aynl * coseo1 + axnl * sineo1 - eo1) / tem5;
			if(fabs(tem5) >= 0.95)

				tem5 = tem5 > 0.0 ? 0.95 : -0.95;
			eo1    = eo1 + tem5;
			ktr = ktr + 1;
		}

		/* ------------- short period preliminary quantities ----------- */
		ecose = axnl*coseo1 + aynl*sineo1;
		esine = axnl*sineo1 - aynl*coseo1;
		el2   = axnl*axnl + aynl*aynl;
		pl    = am*(1.0-el2);
		if (pl < 0.0)
		{
			//         printf("# error pl %f\n", pl);
			satrec[tid].error = 4;
		}
		else
		{
			rl     = am * (1.0 - ecose);
			//rdotl  = sqrt(am) * esine/rl;
			//rvdotl = sqrt(pl) / rl;
			betal  = sqrt(1.0 - el2);
			temp   = esine / (1.0 + betal);
			//sinu   = am / rl * (sineo1 - aynl - axnl * temp);
			//cosu   = am / rl * (coseo1 - axnl + aynl * temp);
			su     = atan2(am / rl * (sineo1 - aynl - axnl * temp), am / rl * (coseo1 - axnl + aynl * temp));
			sin2u  = ( am / rl * (coseo1 - axnl + aynl * temp) +  am / rl * (coseo1 - axnl + aynl * temp)) * (am / rl * (sineo1 - aynl - axnl * temp));
			cos2u  = 1.0 - 2.0 * (am / rl * (sineo1 - aynl - axnl * temp)) * (am / rl * (sineo1 - aynl - axnl * temp));
			temp   = 1.0 / pl;
			temp1  = 0.5 * gravity_constants.j2 * temp;
			temp2  = temp1 * temp;
			//betal  = sqrt(1.0 - el2);
			//temp   = esine / (1.0 + betal);
			//sinu   = am / rl * (sineo1 - aynl - axnl * temp);
			//cosu   = am / rl * (coseo1 - axnl + aynl * temp);
			//su     = atan2(sinu, cosu);
			//sin2u  = (cosu + cosu) * sinu;
			//cos2u  = 1.0 - 2.0 * sinu * sinu;
			//temp   = 1.0 / pl;
			//temp1  = 0.5 * gravity_constants.j2 * temp;
			//temp2  = temp1 * temp;

			/* -------------- update for short period periodics ------------ */
			if (satrec[tid].method == 'd')
			{
				cosisq                 = cosip * cosip;
				satrec[tid].con41  = 3.0*cosisq - 1.0;
				satrec[tid].x1mth2 = 1.0 - cosisq;
				satrec[tid].x7thm1 = 7.0*cosisq - 1.0;
			}
			mrt   = rl * (1.0 - 1.5 * temp2 * betal * satrec[tid].con41) +
				0.5 * temp1 * satrec[tid].x1mth2 * cos2u;
			su    = su - 0.25 * temp2 * satrec[tid].x7thm1 * sin2u;
			xnode = nodep + 1.5 * temp2 * cosip * sin2u;
			xinc  = xincp + 1.5 * temp2 * cosip * sinip * cos2u;
			//mvt   = rdotl - nm * temp1 * satrec[tid].x1mth2 * sin2u / gravity_constants.xke;
			//rvdot = rvdotl + nm * temp1 * (satrec[tid].x1mth2 * cos2u + 1.5 * satrec[tid].con41) / gravity_constants.xke;

			/* --------------------- orientation vectors ------------------- */
			/*sinsu =  sin(su);
			cossu =  cos(su);
			snod  =  sin(xnode);
			cnod  =  cos(xnode);
			sini  =  sin(xinc);
			cosi  =  cos(xinc);*/
			xmx   = -sin(xnode) * cos(xinc);
			xmy   =  cos(xnode) * cos(xinc);
			ux    =  xmx * sin(su) + cos(xnode) * cos(su);
			uy    =  xmy * sin(su) + sin(xnode) * cos(su);
			uz    =  sin(xinc) * sin(su);
			/*sinsu =  sin(su);
			cossu =  cos(su);
			snod  =  sin(xnode);
			cnod  =  cos(xnode);
			sini  =  sin(xinc);
			cosi  =  cos(xinc);
			xmx   = -snod * cosi;
			xmy   =  cnod * cosi;
			ux    =  xmx * sinsu + cnod * cossu;
			uy    =  xmy * sinsu + snod * cossu;
			uz    =  sini * sinsu;*/
			//vx    =  xmx * cossu - cnod * sinsu;
			//vy    =  xmy * cossu - snod * sinsu;
			//vz    =  sini * cossu;

			/* --------- position and velocity (in km and km/sec) ---------- */
			//r[0] = (mrt * ux)* gravity_constants.radiusearthkm;
			//r[1] = (mrt * uy)* gravity_constants.radiusearthkm;
			//r[2] = (mrt * uz)* gravity_constants.radiusearthkm;
			//v[0] = (mvt * ux + rvdot * vx) * vkmpersec;
			//v[1] = (mvt * uy + rvdot * vy) * vkmpersec;
			//v[2] = (mvt * uz + rvdot * vz) * vkmpersec;
			//r[tid].x = (mrt * ux)* gravity_constants.radiusearthkm;
			//r[tid].y = (mrt * uy)* gravity_constants.radiusearthkm;
			//r[tid].z = (mrt * uz)* gravity_constants.radiusearthkm;
			//v[tid].x = (mvt * ux + rvdot * vx) * vkmpersec;
			//v[tid].y = (mvt * uy + rvdot * vy) * vkmpersec;
			//v[tid].z = (mvt * uz + rvdot * vz) * vkmpersec;
			r[tid].x = (mrt * ux);
			r[tid].y = (mrt * uy);
			r[tid].z = (mrt * uz);
			r[tid].w = 1.0;
		}  // if pl > 0

		// sgp4fix for decaying satellites
		if (mrt < 1.0)
		{
			//         printf("# decay condition %11.6f \n",mrt);
			satrec[tid].error = 6;
		}

	}
	////#include "debug7.cpp"
	//return satrec[tid].error;
}  // end sgp4

__device__ static void dspace
	(
	int irez,
	double d2201,  double d2211,  double d3210,   double d3222,  double d4410,
	double d4422,  double d5220,  double d5232,   double d5421,  double d5433,
	double dedt,   double del1,   double del2,    double del3,   double didt,
	double dmdt,   double dnodt,  double domdt,   double argpo,  double argpdot,
	double t,      double tc,     double gsto,    double xfact,  double xlamo,
	double no,
	double& atime, double& em,    double& argpm,  double& inclm, double& xli,
	double& mm,    double& xni,   double& nodem,  double& dndt,  double& nm
	)
{
	//const double twopi = 2.0 * HIP_PI;
	int iretn , iret;
	double delt, ft, theta, x2li, x2omi, xl, xldot , xnddt, xndt, xomi, g22, g32,
		g44, g52, g54, fasx2, fasx4, fasx6, rptim , step2, stepn , stepp;

	ft    = 0.0;
	fasx2 = 0.13130908;
	fasx4 = 2.8843198;
	fasx6 = 0.37448087;
	g22   = 5.7686396;
	g32   = 0.95240898;
	g44   = 1.8014998;
	g52   = 1.0508330;
	g54   = 4.4108898;
	rptim = 4.37526908801129966e-3; // this equates to 7.29211514668855e-5 rad/sec
	stepp =    720.0;
	stepn =   -720.0;
	step2 = 259200.0;

	/* ----------- calculate deep space resonance effects ----------- */
	dndt   = 0.0;
	theta  = fmod(gsto + tc * rptim, 2.0 * HIP_PI);
	em     = em + dedt * t;

	inclm  = inclm + didt * t;
	argpm  = argpm + domdt * t;
	nodem  = nodem + dnodt * t;
	mm     = mm + dmdt * t;

	//   sgp4fix for negative inclinations
	//   the following if statement should be commented out
	//  if (inclm < 0.0)
	// {
	//    inclm = -inclm;
	//    argpm = argpm - pi;
	//    nodem = nodem + pi;
	//  }

	/* - update resonances : numerical (euler-maclaurin) integration - */
	/* ------------------------- epoch restart ----------------------  */
	//   sgp4fix for propagator problems
	//   the following integration works for negative time steps and periods
	//   the specific changes are unknown because the original code was so convoluted

	ft    = 0.0;
	atime = 0.0;
	if (irez != 0)
	{
		if ((atime == 0.0) || ((t >= 0.0) && (atime < 0.0)) ||
			((t < 0.0) && (atime >= 0.0)))
		{
			if (t >= 0.0)
				delt = stepp;
			else
				delt = stepn;
			atime  = 0.0;
			xni    = no;
			xli    = xlamo;
		}
		iretn = 381; // added for do loop
		iret  =   0; // added for loop
		while (iretn == 381)
		{
			if ((fabs(t) < fabs(atime)) || (iret == 351))
			{
				if (t >= 0.0)
					delt = stepn;
				else
					delt = stepp;
				iret  = 351;
				iretn = 381;
			}
			else
			{
				if (t > 0.0)  // error if prev if has atime:=0.0 and t:=0.0 (ge)
					delt = stepp;
				else
					delt = stepn;
				if (fabs(t - atime) >= stepp)
				{
					iret  = 0;
					iretn = 381;
				}
				else
				{
					ft    = t - atime;
					iretn = 0;
				}
			}

			/* ------------------- dot terms calculated ------------- */
			/* ----------- near - synchronous resonance terms ------- */
			if (irez != 2)
			{
				xndt  = del1 * sin(xli - fasx2) + del2 * sin(2.0 * (xli - fasx4)) +
					del3 * sin(3.0 * (xli - fasx6));
				xldot = xni + xfact;
				xnddt = del1 * cos(xli - fasx2) +
					2.0 * del2 * cos(2.0 * (xli - fasx4)) +
					3.0 * del3 * cos(3.0 * (xli - fasx6));
				xnddt = xnddt * xldot;
			}
			else
			{
				/* --------- near - half-day resonance terms -------- */
				xomi  = argpo + argpdot * atime;
				x2omi = xomi + xomi;
				x2li  = xli + xli;
				xndt  = d2201 * sin(x2omi + xli - g22) + d2211 * sin(xli - g22) +
					d3210 * sin(xomi + xli - g32)  + d3222 * sin(-xomi + xli - g32)+
					d4410 * sin(x2omi + x2li - g44)+ d4422 * sin(x2li - g44) +
					d5220 * sin(xomi + xli - g52)  + d5232 * sin(-xomi + xli - g52)+
					d5421 * sin(xomi + x2li - g54) + d5433 * sin(-xomi + x2li - g54);
				xldot = xni + xfact;
				xnddt = d2201 * cos(x2omi + xli - g22) + d2211 * cos(xli - g22) +
					d3210 * cos(xomi + xli - g32) + d3222 * cos(-xomi + xli - g32) +
					d5220 * cos(xomi + xli - g52) + d5232 * cos(-xomi + xli - g52) +
					2.0 * (d4410 * cos(x2omi + x2li - g44) +
					d4422 * cos(x2li - g44) + d5421 * cos(xomi + x2li - g54) +
					d5433 * cos(-xomi + x2li - g54));
				xnddt = xnddt * xldot;
			}

			/* ----------------------- integrator ------------------- */
			if (iretn == 381)
			{
				xli   = xli + xldot * delt + xndt * step2;
				xni   = xni + xndt * delt + xnddt * step2;
				atime = atime + delt;
			}
		}  // while iretn = 381

		nm = xni + xndt * ft + xnddt * ft * ft * 0.5;
		xl = xli + xldot * ft + xndt * ft * ft * 0.5;
		if (irez != 1)
		{
			mm   = xl - 2.0 * nodem + 2.0 * theta;
			dndt = nm - no;
		}
		else
		{
			mm   = xl - nodem - argpm + theta;
			dndt = nm - no;
		}
		nm = no + dndt;
	}

	//#include "debug4.cpp"
}  // end dsspace
