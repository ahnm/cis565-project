#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2011 NVIDIA Corporation.  All rights reserved.
 *
 * NVIDIA Corporation and its licensors retain all intellectual property and 
 * proprietary rights in and to this software and related documentation. 
 * Any use, reproduction, disclosure, or distribution of this software 
 * and related documentation without an express license agreement from
 * NVIDIA Corporation is strictly prohibited.
 *
 * Please refer to the applicable NVIDIA end user license agreement (EULA) 
 * associated with this source code for terms and conditions that govern 
 * your use of this NVIDIA software.
 * 
 */

#include <hip/hip_math_constants.h>

#include "common.h"
#include "constants.h"
#include "commonCUDA.cuh"
#include "satelliterecord.h"
#include "functionsKernel.cu"


//extern __device__ __constant__ gravconstant_t gravity_constants;
__device__ static void dscom
	(
	double epoch,  double ep,     double argpp,   double tc,     double inclp,
	double nodep,  double np,
	double& snodm, double& cnodm, double& sinim,  double& cosim, double& sinomm,
	double& cosomm,double& day,   double& e3,     double& ee2,   double& em,
	double& emsq,  double& gam,   double& peo,    double& pgho,  double& pho,
	double& pinco, double& plo,   double& rtemsq, double& se2,   double& se3,
	double& sgh2,  double& sgh3,  double& sgh4,   double& sh2,   double& sh3,
	double& si2,   double& si3,   double& sl2,    double& sl3,   double& sl4,
	double& s1,    double& s2,    double& s3,     double& s4,    double& s5,
	double& s6,    double& s7,    double& ss1,    double& ss2,   double& ss3,
	double& ss4,   double& ss5,   double& ss6,    double& ss7,   double& sz1,
	double& sz2,   double& sz3,   double& sz11,   double& sz12,  double& sz13,
	double& sz21,  double& sz22,  double& sz23,   double& sz31,  double& sz32,
	double& sz33,  double& xgh2,  double& xgh3,   double& xgh4,  double& xh2,
	double& xh3,   double& xi2,   double& xi3,    double& xl2,   double& xl3,
	double& xl4,   double& nm,    double& z1,     double& z2,    double& z3,
	double& z11,   double& z12,   double& z13,    double& z21,   double& z22,
	double& z23,   double& z31,   double& z32,    double& z33,   double& zmol,
	double& zmos
	);

__device__ static void dpper
	(
	double e3,     double ee2,    double peo,     double pgho,   double pho,
	double pinco,  double plo,    double se2,     double se3,    double sgh2,
	double sgh3,   double sgh4,   double sh2,     double sh3,    double si2,
	double si3,    double sl2,    double sl3,     double sl4,    double t,
	double xgh2,   double xgh3,   double xgh4,    double xh2,    double xh3,
	double xi2,    double xi3,    double xl2,     double xl3,    double xl4,
	double zmol,   double zmos,   double inclo,
	char init,
	double& ep,    double& inclp, double& nodep,  double& argpp, double& mp
	);
__device__ static void dsinit
	(
	double cosim,  double emsq,   double argpo,   double s1,     double s2,
	double s3,     double s4,     double s5,      double sinim,  double ss1,
	double ss2,    double ss3,    double ss4,     double ss5,    double sz1,
	double sz3,    double sz11,   double sz13,    double sz21,   double sz23,
	double sz31,   double sz33,   double t,       double tc,     double gsto,
	double mo,     double mdot,   double no,      double nodeo,  double nodedot,
	double xpidot, double z1,     double z3,      double z11,    double z13,
	double z21,    double z23,    double z31,     double z33,    double ecco,
	double eccsq,  double& em,    double& argpm,  double& inclm, double& mm,
	double& nm,    double& nodem,
	int& irez,
	double& atime, double& d2201, double& d2211,  double& d3210, double& d3222,
	double& d4410, double& d4422, double& d5220,  double& d5232, double& d5421,
	double& d5433, double& dedt,  double& didt,   double& dmdt,  double& dndt,
	double& dnodt, double& domdt, double& del1,   double& del2,  double& del3,
	double& xfact, double& xlamo, double& xli,    double& xni
	);

__global__ void sgp4initkernel(satelliterecord_soa_t *satrec, int N)
{
#define STRIDE		32
#define OFFSET		0
#define GROUP_SIZE	512
	int block_start_idx = blockIdx.x * blockDim.x;
	//int tid = block_start_idx + ((threadIdx.x + OFFSET) % STRIDE);
	int tid = block_start_idx + threadIdx.x;
	if(tid < N){
		/* --------------------- local variables ------------------------ */
		double ao, ainv,   con42, cosio, sinio, cosio2, eccsq,
			omeosq, posq,   rp,     rteosq,
			cnodm , snodm , cosim , sinim , cosomm, sinomm, cc1sq ,
			cc2   , cc3   , coef  , coef1 , cosio4, day   , dndt  ,
			em    , emsq  , eeta  , etasq , gam   , argpm , nodem ,
			inclm , mm    , nm    , perige, pinvsq, psisq , qzms24,
			rtemsq, s1    , s2    , s3    , s4    , s5    , s6    ,
			s7    , sfour , ss1   , ss2   , ss3   , ss4   , ss5   ,
			ss6   , ss7   , sz1   , sz2   , sz3   , sz11  , sz12  ,
			sz13  , sz21  , sz22  , sz23  , sz31  , sz32  , sz33  ,
			tc    , temp  , temp1 , temp2 , temp3 , tsi   , xpidot,
			xhdot1, z1    , z2    , z3    , z11   , z12   , z13   , 
			z21   , z22   , z23   , z31   , z32   , z33,
			qzms2t, ss, /*j2, j3oj2, j4, */x2o3/*, r[3], v[3]*/
			/*,tumin, mu, radiusearthkm, xke, j3*/;

		/* ------------------------ initialization --------------------- */
		// sgp4fix divisor for divide by zero check on inclination
		const double temp4    =   1.0 + cos(HIP_PI-1.0e-9);
		/* ----------- set all near earth variables to zero ------------ */
		satrec[tid].isimp   = 0;   satrec[tid].method = 'n'; satrec[tid].aycof    = 0.0;
		satrec[tid].con41   = 0.0; satrec[tid].cc1    = 0.0; satrec[tid].cc4      = 0.0;
		satrec[tid].cc5     = 0.0; satrec[tid].d2     = 0.0; satrec[tid].d3       = 0.0;
		satrec[tid].d4      = 0.0; satrec[tid].delmo  = 0.0; satrec[tid].eta      = 0.0;
		satrec[tid].argpdot = 0.0; satrec[tid].omgcof = 0.0; satrec[tid].sinmao   = 0.0;
		satrec[tid].t       = 0.0; satrec[tid].t2cof  = 0.0; satrec[tid].t3cof    = 0.0;
		satrec[tid].t4cof   = 0.0; satrec[tid].t5cof  = 0.0; satrec[tid].x1mth2   = 0.0;
		satrec[tid].x7thm1  = 0.0; satrec[tid].mdot   = 0.0; satrec[tid].nodedot  = 0.0;
		satrec[tid].xlcof   = 0.0; satrec[tid].xmcof  = 0.0; satrec[tid].nodecf   = 0.0;

		/* ----------- set all deep space variables to zero ------------ */
		satrec[tid].irez  = 0;   satrec[tid].d2201 = 0.0; satrec[tid].d2211 = 0.0;
		satrec[tid].d3210 = 0.0; satrec[tid].d3222 = 0.0; satrec[tid].d4410 = 0.0;
		satrec[tid].d4422 = 0.0; satrec[tid].d5220 = 0.0; satrec[tid].d5232 = 0.0;
		satrec[tid].d5421 = 0.0; satrec[tid].d5433 = 0.0; satrec[tid].dedt  = 0.0;
		satrec[tid].del1  = 0.0; satrec[tid].del2  = 0.0; satrec[tid].del3  = 0.0;
		satrec[tid].didt  = 0.0; satrec[tid].dmdt  = 0.0; satrec[tid].dnodt = 0.0;
		satrec[tid].domdt = 0.0; satrec[tid].e3    = 0.0; satrec[tid].ee2   = 0.0;
		satrec[tid].peo   = 0.0; satrec[tid].pgho  = 0.0; satrec[tid].pho   = 0.0;
		satrec[tid].pinco = 0.0; satrec[tid].plo   = 0.0; satrec[tid].se2   = 0.0;
		satrec[tid].se3   = 0.0; satrec[tid].sgh2  = 0.0; satrec[tid].sgh3  = 0.0;
		satrec[tid].sgh4  = 0.0; satrec[tid].sh2   = 0.0; satrec[tid].sh3   = 0.0;
		satrec[tid].si2   = 0.0; satrec[tid].si3   = 0.0; satrec[tid].sl2   = 0.0;
		satrec[tid].sl3   = 0.0; satrec[tid].sl4   = 0.0; satrec[tid].gsto  = 0.0;
		satrec[tid].xfact = 0.0; satrec[tid].xgh2  = 0.0; satrec[tid].xgh3  = 0.0;
		satrec[tid].xgh4  = 0.0; satrec[tid].xh2   = 0.0; satrec[tid].xh3   = 0.0;
		satrec[tid].xi2   = 0.0; satrec[tid].xi3   = 0.0; satrec[tid].xl2   = 0.0;
		satrec[tid].xl3   = 0.0; satrec[tid].xl4   = 0.0; satrec[tid].xlamo = 0.0;
		satrec[tid].zmol  = 0.0; satrec[tid].zmos  = 0.0; satrec[tid].atime = 0.0;
		satrec[tid].xli   = 0.0; satrec[tid].xni   = 0.0;

		// sgp4fix - note the following variables are also passed directly via satrec.
		// it is possible to streamline the sgp4init call by deleting the "x"
		// variables, but the user would need to set the satrec.* values first. we
		// include the additional assignments in case twoline2rv is not used.
		satrec[tid].method = 'n';
		//satrec->bstar[tid]   = xbstar;
		//satrec->ecco[tid]    = xecco;
		//satrec->argpo[tid]   = xargpo;
		//satrec->inclo[tid]   = xinclo;
		//satrec->mo[tid]	    = xmo;
		//satrec->no[tid]	    = xno;
		//satrec->nodeo[tid]   = xnodeo;

		/* ------------------------ earth constants ----------------------- */
		// sgp4fix identify constants and allow alternate values
		//getgravconst( whichconst, tumin, mu, radiusearthkm, xke, j2, j3, j4, j3oj2 );
	
		ss     = 78.0 / gravity_constants.radiusearthkm + 1.0;
		qzms2t = pow(((120.0 - 78.0) / gravity_constants.radiusearthkm), 4);
		x2o3   =  2.0 / 3.0;

		satrec[tid].init = 'y';
		satrec[tid].t	 = 0.0;
		//initl
		//	(
		//	satn, whichconst, satrec->ecco[tid], epoch, satrec->inclo[tid], satrec->no[tid], satrec->method[tid],
		//	ainv, ao, satrec->con41[tid], con42, cosio, cosio2, eccsq, omeosq,
		//	posq, rp, rteosq, sinio, satrec->gsto[tid]
		//	);
		eccsq = pow(satrec[tid].ecco, 2.0);
		omeosq = 1.0 - eccsq;
		rteosq = sqrt(omeosq);
		cosio  = cos(satrec[tid].inclo);
		cosio2 = cosio * cosio;
		double ak    = pow(gravity_constants.xke / satrec[tid].no, 2.0/3.0);
		double d1    = 0.75 * gravity_constants.j2 * (3.0 * cosio2 - 1.0) / (rteosq * omeosq);
		double del   = d1 / pow(gravity_constants.xke / satrec[tid].no, 4.0/3.0);
		double adel  = ak * (1.0 - del * del - del * (1.0 / 3.0 + 134.0 * del * del / 81.0));
		del   = d1/(adel * adel);
		satrec[tid].no    = satrec[tid].no / (1.0 + del);

		ao    = pow(gravity_constants.xke / satrec[tid].no, x2o3);
		sinio = sin(satrec[tid].inclo);
		double po    = ao * omeosq;
		con42 = 1.0 - 5.0 * cosio2;
		satrec[tid].con41 = -con42-cosio2-cosio2;
		ainv  = 1.0 / ao;
		posq  = po * po;
		rp    = ao * (1.0 -satrec[tid].ecco);

		satrec[tid].gsto = gstime(satrec[tid].jdsatepoch);

		satrec[tid].error = 0;

		if (rp < 1.0)
		{
			//         printf("# *** satn%d epoch elts sub-orbital ***\n", satn);
			satrec[tid].error = 5;
		}

		if ((omeosq >= 0.0 ) || ( satrec[tid].no >= 0.0))
		{
			satrec[tid].isimp = 0;
			if (rp < (220.0 / gravity_constants.radiusearthkm + 1.0))
				satrec[tid].isimp = 1;
			sfour  = ss;
			qzms24 = qzms2t;
			perige = (rp - 1.0) * gravity_constants.radiusearthkm;

			/* - for perigees below 156 km, s and qoms2t are altered - */
			if (perige < 156.0)
			{
				sfour = perige - 78.0;
				if (perige < 98.0)
					sfour = 20.0;
				qzms24 = pow(((120.0 - sfour) / gravity_constants.radiusearthkm), 4.0);
				sfour  = sfour / gravity_constants.radiusearthkm + 1.0;
			}
			pinvsq = 1.0 / posq;

			tsi  = 1.0 / (ao - sfour);
			satrec[tid].eta  = ao * satrec[tid].ecco * tsi;
			etasq = satrec[tid].eta * satrec[tid].eta;
			eeta  = satrec[tid].ecco * satrec[tid].eta;
			psisq = fabs(1.0 - etasq);
			coef  = qzms24 * pow(tsi, 4.0);
			coef1 = coef / pow(psisq, 3.5);
			cc2   = coef1 * satrec[tid].no * (ao * (1.0 + 1.5 * etasq + eeta *
				(4.0 + etasq)) + 0.375 * gravity_constants.j2 * tsi / psisq * satrec[tid].con41 *
				(8.0 + 3.0 * etasq * (8.0 + etasq)));
			satrec[tid].cc1   = satrec[tid].bstar * cc2;
			cc3   = 0.0;
			if (satrec[tid].ecco > 1.0e-4)
				cc3 = -2.0 * coef * tsi * gravity_constants.j3oj2 * satrec[tid].no * sinio / satrec[tid].ecco;
			satrec[tid].x1mth2 = 1.0 - cosio2;
			satrec[tid].cc4    = 2.0* satrec[tid].no * coef1 * ao * omeosq *
				(satrec[tid].eta * (2.0 + 0.5 * etasq) + satrec[tid].ecco *
				(0.5 + 2.0 * etasq) - gravity_constants.j2 * tsi / (ao * psisq) *
				(-3.0 * satrec[tid].con41 * (1.0 - 2.0 * eeta + etasq *
				(1.5 - 0.5 * eeta)) + 0.75 * satrec[tid].x1mth2 *
				(2.0 * etasq - eeta * (1.0 + etasq)) * cos(2.0 * satrec[tid].argpo)));
			satrec[tid].cc5 = 2.0 * coef1 * ao * omeosq * (1.0 + 2.75 *
				(etasq + eeta) + eeta * etasq);
			cosio4 = cosio2 * cosio2;
			//temp1  = 1.5 * j2 * pinvsq * satrec.no;
			temp2  = 0.5 * 1.5 * gravity_constants.j2 * pinvsq * satrec[tid].no * gravity_constants.j2 * pinvsq;
			temp3  = -0.46875 * gravity_constants.j4 * pinvsq * pinvsq * satrec[tid].no;
			satrec[tid].mdot     = satrec[tid].no + 0.5 * 1.5 * gravity_constants.j2 * pinvsq * satrec[tid].no * rteosq * satrec[tid].con41 + 0.0625 *
				temp2 * rteosq * (13.0 - 78.0 * cosio2 + 137.0 * cosio4);
			satrec[tid].argpdot  = -0.5 * 1.5 * gravity_constants.j2 * pinvsq * satrec[tid].no * con42 + 0.0625 * temp2 *
				(7.0 - 114.0 * cosio2 + 395.0 * cosio4) +
				temp3 * (3.0 - 36.0 * cosio2 + 49.0 * cosio4);
			xhdot1            = -1.5 * gravity_constants.j2 * pinvsq * satrec[tid].no * cosio;
			satrec[tid].nodedot = xhdot1 + (0.5 * temp2 * (4.0 - 19.0 * cosio2) +
				2.0 * temp3 * (3.0 - 7.0 * cosio2)) * cosio;
			xpidot            =  satrec[tid].argpdot + satrec[tid].nodedot;
			satrec[tid].omgcof   = satrec[tid].bstar * cc3 * cos(satrec[tid].argpo);
			satrec[tid].xmcof    = 0.0;
			if (satrec[tid].ecco > 1.0e-4)
				satrec[tid].xmcof = -x2o3 * coef * satrec[tid].bstar / eeta;
			satrec[tid].nodecf = 3.5 * omeosq * xhdot1 * satrec[tid].cc1;
			satrec[tid].t2cof   = 1.5 * satrec[tid].cc1;
			// sgp4fix for divide by zero with xinco = 180 deg
			if (fabs(cosio+1.0) > 1.5e-12)
				satrec[tid].xlcof = -0.25 * gravity_constants.j3oj2 * sinio * (3.0 + 5.0 * cosio) / (1.0 + cosio);
			else
				satrec[tid].xlcof = -0.25 * gravity_constants.j3oj2 * sinio * (3.0 + 5.0 * cosio) / temp4;
			satrec[tid].aycof   = -0.5 * gravity_constants.j3oj2 * sinio;
			satrec[tid].delmo   = pow((1.0 + satrec[tid].eta * cos(satrec[tid].mo)), 3);
			satrec[tid].sinmao  = sin(satrec[tid].mo);
			satrec[tid].x7thm1  = 7.0 * cosio2 - 1.0;

			/* --------------- deep space initialization ------------- */
			if ((2*HIP_PI / satrec[tid].no) >= 225.0)
			{
				satrec[tid].method = 'd';
				satrec[tid].isimp  = 1;
				tc    =  0.0;
				inclm = satrec[tid].inclo;

				dscom
					(
					satrec[tid].jdsatepoch-2433281.5, satrec[tid].ecco, satrec[tid].argpo, tc, satrec[tid].inclo, satrec[tid].nodeo,
					satrec[tid].no, snodm, cnodm,  sinim, cosim,sinomm,     cosomm,
					day, satrec[tid].e3, satrec[tid].ee2, em,         emsq, gam,
					satrec[tid].peo,  satrec[tid].pgho,   satrec[tid].pho, satrec[tid].pinco,
					satrec[tid].plo,  rtemsq,        satrec[tid].se2, satrec[tid].se3,
					satrec[tid].sgh2, satrec[tid].sgh3,   satrec[tid].sgh4,
					satrec[tid].sh2,  satrec[tid].sh3,    satrec[tid].si2, satrec[tid].si3,
					satrec[tid].sl2,  satrec[tid].sl3,    satrec[tid].sl4, s1, s2, s3, s4, s5,
					s6,   s7,   ss1,  ss2,  ss3,  ss4,  ss5,  ss6,  ss7, sz1, sz2, sz3,
					sz11, sz12, sz13, sz21, sz22, sz23, sz31, sz32, sz33,
					satrec[tid].xgh2, satrec[tid].xgh3,   satrec[tid].xgh4, satrec[tid].xh2,
					satrec[tid].xh3,  satrec[tid].xi2,    satrec[tid].xi3,  satrec[tid].xl2,
					satrec[tid].xl3,  satrec[tid].xl4,    nm, z1, z2, z3, z11,
					z12, z13, z21, z22, z23, z31, z32, z33,
					satrec[tid].zmol, satrec[tid].zmos
					);
				dpper
					(
					satrec[tid].e3, satrec[tid].ee2, satrec[tid].peo, satrec[tid].pgho,
					satrec[tid].pho, satrec[tid].pinco, satrec[tid].plo, satrec[tid].se2,
					satrec[tid].se3, satrec[tid].sgh2, satrec[tid].sgh3, satrec[tid].sgh4,
					satrec[tid].sh2, satrec[tid].sh3, satrec[tid].si2,  satrec[tid].si3,
					satrec[tid].sl2, satrec[tid].sl3, satrec[tid].sl4,  satrec[tid].t,
					satrec[tid].xgh2,satrec[tid].xgh3,satrec[tid].xgh4, satrec[tid].xh2,
					satrec[tid].xh3, satrec[tid].xi2, satrec[tid].xi3,  satrec[tid].xl2,
					satrec[tid].xl3, satrec[tid].xl4, satrec[tid].zmol, satrec[tid].zmos, inclm, satrec[tid].init,
					satrec[tid].ecco, satrec[tid].inclo, satrec[tid].nodeo, satrec[tid].argpo, satrec[tid].mo
					);

				argpm  = 0.0;
				nodem  = 0.0;
				mm     = 0.0;

				dsinit
					(
					cosim, emsq, satrec[tid].argpo, s1, s2, s3, s4, s5, sinim, ss1, ss2, ss3, ss4,
					ss5, sz1, sz3, sz11, sz13, sz21, sz23, sz31, sz33, satrec[tid].t, tc,
					satrec[tid].gsto, satrec[tid].mo, satrec[tid].mdot, satrec[tid].no, satrec[tid].nodeo,
					satrec[tid].nodedot, xpidot, z1, z3, z11, z13, z21, z23, z31, z33,
					satrec[tid].ecco, eccsq, em, argpm, inclm, mm, nm, nodem,
					satrec[tid].irez,  satrec[tid].atime,
					satrec[tid].d2201, satrec[tid].d2211, satrec[tid].d3210, satrec[tid].d3222 ,
					satrec[tid].d4410, satrec[tid].d4422, satrec[tid].d5220, satrec[tid].d5232,
					satrec[tid].d5421, satrec[tid].d5433, satrec[tid].dedt,  satrec[tid].didt,
					satrec[tid].dmdt,  dndt,         satrec[tid].dnodt, satrec[tid].domdt ,
					satrec[tid].del1,  satrec[tid].del2,  satrec[tid].del3,  satrec[tid].xfact,
					satrec[tid].xlamo, satrec[tid].xli,   satrec[tid].xni
					);
			}

			/* ----------- set variables if not deep space ----------- */
			if (satrec[tid].isimp != 1)
			{
				cc1sq          = satrec[tid].cc1 * satrec[tid].cc1;
				satrec[tid].d2    = 4.0 * ao * tsi * cc1sq;
				temp           = satrec[tid].d2 * tsi * satrec[tid].cc1 / 3.0;
				satrec[tid].d3    = (17.0 * ao + sfour) * temp;
				satrec[tid].d4    = 0.5 * temp * ao * tsi * (221.0 * ao + 31.0 * sfour) *
					satrec[tid].cc1;
				satrec[tid].t3cof = satrec[tid].d2 + 2.0 * cc1sq;
				satrec[tid].t4cof = 0.25 * (3.0 * satrec[tid].d3 + satrec[tid].cc1 *
					(12.0 * satrec[tid].d2 + 10.0 * cc1sq));
				satrec[tid].t5cof = 0.2 * (3.0 * satrec[tid].d4 +
					12.0 * satrec[tid].cc1 * satrec[tid].d3 +
					6.0 * satrec[tid].d2 * satrec[tid].d2 +
					15.0 * cc1sq * (2.0 * satrec[tid].d2 + cc1sq));
			}
		} // if omeosq = 0 ...

		/* finally propogate to zero epoch to initialise all others. */
		if(satrec[tid].error == 0)
			//sgp4(whichconst, satrec, 0.0, r, v);

		satrec[tid].init = 'n';

	
	}
	//#include "debug6.cpp"
	//return satrec.error;
}  // end sgp4init


/////////////////////////////////////////////////////////////////////////////////
///// \brief add two vectors of size _count_
/////
///// CUDA kernel
///// \param[in]  op1   term one
///// \param[in]  op2   term two
///// \param[in]  count vector size
///// \param[out] sum   result
/////////////////////////////////////////////////////////////////////////////////
//__global__ 
//void AddKernel(const float *op1, const float *op2, int count, float *sum)
//{
//    const int pos = threadIdx.x + blockIdx.x * blockDim.x;
//
//    if (pos >= count) return;
//
//    sum[pos] = op1[pos] + op2[pos];
//}
//
/////////////////////////////////////////////////////////////////////////////////
///// \brief add two vectors of size _count_
///// \param[in]  op1   term one
///// \param[in]  op2   term two
///// \param[in]  count vector size
///// \param[out] sum   result
/////////////////////////////////////////////////////////////////////////////////
//static
//void Add(const float *op1, const float *op2, int count, float *sum)
//{
//    dim3 threads(256);
//    dim3 blocks(iDivUp(count, threads.x));
//
//    AddKernel<<<blocks, threads>>>(op1, op2, count, sum);
//}



/*-----------------------------------------------------------------------------
*
*                           procedure dscom
*
*  this procedure provides deep space common items used by both the secular
*    and periodics subroutines.  input is provided as shown. this routine
*    used to be called dpper, but the functions inside weren't well organized.
*
*  author        : david vallado                  719-573-2600   28 jun 2005
*
*  inputs        :
*    epoch       -
*    ep          - eccentricity
*    argpp       - argument of perigee
*    tc          -
*    inclp       - inclination
*    nodep       - right ascension of ascending node
*    np          - mean motion
*
*  outputs       :
*    sinim  , cosim  , sinomm , cosomm , snodm  , cnodm
*    day         -
*    e3          -
*    ee2         -
*    em          - eccentricity
*    emsq        - eccentricity squared
*    gam         -
*    peo         -
*    pgho        -
*    pho         -
*    pinco       -
*    plo         -
*    rtemsq      -
*    se2, se3         -
*    sgh2, sgh3, sgh4        -
*    sh2, sh3, si2, si3, sl2, sl3, sl4         -
*    s1, s2, s3, s4, s5, s6, s7          -
*    ss1, ss2, ss3, ss4, ss5, ss6, ss7, sz1, sz2, sz3         -
*    sz11, sz12, sz13, sz21, sz22, sz23, sz31, sz32, sz33        -
*    xgh2, xgh3, xgh4, xh2, xh3, xi2, xi3, xl2, xl3, xl4         -
*    nm          - mean motion
*    z1, z2, z3, z11, z12, z13, z21, z22, z23, z31, z32, z33         -
*    zmol        -
*    zmos        -
*
*  locals        :
*    a1, a2, a3, a4, a5, a6, a7, a8, a9, a10         -
*    betasq      -
*    cc          -
*    ctem, stem        -
*    x1, x2, x3, x4, x5, x6, x7, x8          -
*    xnodce      -
*    xnoi        -
*    zcosg  , zsing  , zcosgl , zsingl , zcosh  , zsinh  , zcoshl , zsinhl ,
*    zcosi  , zsini  , zcosil , zsinil ,
*    zx          -
*    zy          -
*
*  coupling      :
*    none.
*
*  references    :
*    hoots, roehrich, norad spacetrack report #3 1980
*    hoots, norad spacetrack report #6 1986
*    hoots, schumacher and glover 2004
*    vallado, crawford, hujsak, kelso  2006
----------------------------------------------------------------------------*/

__device__ static void dscom
	(
	double epoch,  double ep,     double argpp,   double tc,     double inclp,
	double nodep,  double np,
	double& snodm, double& cnodm, double& sinim,  double& cosim, double& sinomm,
	double& cosomm,double& day,   double& e3,     double& ee2,   double& em,
	double& emsq,  double& gam,   double& peo,    double& pgho,  double& pho,
	double& pinco, double& plo,   double& rtemsq, double& se2,   double& se3,
	double& sgh2,  double& sgh3,  double& sgh4,   double& sh2,   double& sh3,
	double& si2,   double& si3,   double& sl2,    double& sl3,   double& sl4,
	double& s1,    double& s2,    double& s3,     double& s4,    double& s5,
	double& s6,    double& s7,    double& ss1,    double& ss2,   double& ss3,
	double& ss4,   double& ss5,   double& ss6,    double& ss7,   double& sz1,
	double& sz2,   double& sz3,   double& sz11,   double& sz12,  double& sz13,
	double& sz21,  double& sz22,  double& sz23,   double& sz31,  double& sz32,
	double& sz33,  double& xgh2,  double& xgh3,   double& xgh4,  double& xh2,
	double& xh3,   double& xi2,   double& xi3,    double& xl2,   double& xl3,
	double& xl4,   double& nm,    double& z1,     double& z2,    double& z3,
	double& z11,   double& z12,   double& z13,    double& z21,   double& z22,
	double& z23,   double& z31,   double& z32,    double& z33,   double& zmol,
	double& zmos
	)
{
	/* -------------------------- constants ------------------------- */
	const double zes     =  0.01675;
	const double zel     =  0.05490;
	const double c1ss    =  2.9864797e-6;
	const double c1l     =  4.7968065e-7;
	const double zsinis  =  0.39785416;
	const double zcosis  =  0.91744867;
	const double zcosgs  =  0.1945905;
	const double zsings  = -0.98088458;
	const double twopi   =  2.0 * HIP_PI;

	/* --------------------- local variables ------------------------ */
	int lsflg;
	double a1    , a2    , a3    , a4    , a5    , a6    , a7    ,
		a8    , a9    , a10   , betasq, cc    , ctem  , stem  ,
		x1    , x2    , x3    , x4    , x5    , x6    , x7    ,
		x8    , xnodce, xnoi  , zcosg , zcosgl, zcosh , zcoshl,
		zcosi , zcosil, zsing , zsingl, zsinh , zsinhl, zsini ,
		zsinil, zx    , zy;

	nm     = np;
	em     = ep;
	snodm  = sin(nodep);
	cnodm  = cos(nodep);
	sinomm = sin(argpp);
	cosomm = cos(argpp);
	sinim  = sin(inclp);
	cosim  = cos(inclp);
	emsq   = em * em;
	betasq = 1.0 - emsq;
	rtemsq = sqrt(betasq);

	/* ----------------- initialize lunar solar terms --------------- */
	peo    = 0.0;
	pinco  = 0.0;
	plo    = 0.0;
	pgho   = 0.0;
	pho    = 0.0;
	day    = epoch + 18261.5 + tc / 1440.0;
	xnodce = fmod(4.5236020 - 9.2422029e-4 * day, twopi);
	stem   = sin(xnodce);
	ctem   = cos(xnodce);
	zcosil = 0.91375164 - 0.03568096 * ctem;
	zsinil = sqrt(1.0 - zcosil * zcosil);
	zsinhl = 0.089683511 * stem / zsinil;
	zcoshl = sqrt(1.0 - zsinhl * zsinhl);
	gam    = 5.8351514 + 0.0019443680 * day;
	zx     = 0.39785416 * stem / zsinil;
	zy     = zcoshl * ctem + 0.91744867 * zsinhl * stem;
	zx     = atan2(zx, zy);
	zx     = gam + zx - xnodce;
	zcosgl = cos(zx);
	zsingl = sin(zx);

	/* ------------------------- do solar terms --------------------- */
	zcosg = zcosgs;
	zsing = zsings;
	zcosi = zcosis;
	zsini = zsinis;
	zcosh = cnodm;
	zsinh = snodm;
	cc    = c1ss;
	xnoi  = 1.0 / nm;

	for (lsflg = 1; lsflg <= 2; lsflg++)
	{
		a1  =   zcosg * zcosh + zsing * zcosi * zsinh;
		a3  =  -zsing * zcosh + zcosg * zcosi * zsinh;
		a7  =  -zcosg * zsinh + zsing * zcosi * zcosh;
		a8  =   zsing * zsini;
		a9  =   zsing * zsinh + zcosg * zcosi * zcosh;
		a10 =   zcosg * zsini;
		a2  =   cosim * a7 + sinim * a8;
		a4  =   cosim * a9 + sinim * a10;
		a5  =  -sinim * a7 + cosim * a8;
		a6  =  -sinim * a9 + cosim * a10;

		x1  =  a1 * cosomm + a2 * sinomm;
		x2  =  a3 * cosomm + a4 * sinomm;
		x3  = -a1 * sinomm + a2 * cosomm;
		x4  = -a3 * sinomm + a4 * cosomm;
		x5  =  a5 * sinomm;
		x6  =  a6 * sinomm;
		x7  =  a5 * cosomm;
		x8  =  a6 * cosomm;

		z31 = 12.0 * x1 * x1 - 3.0 * x3 * x3;
		z32 = 24.0 * x1 * x2 - 6.0 * x3 * x4;
		z33 = 12.0 * x2 * x2 - 3.0 * x4 * x4;
		z1  =  3.0 *  (a1 * a1 + a2 * a2) + z31 * emsq;
		z2  =  6.0 *  (a1 * a3 + a2 * a4) + z32 * emsq;
		z3  =  3.0 *  (a3 * a3 + a4 * a4) + z33 * emsq;
		z11 = -6.0 * a1 * a5 + emsq *  (-24.0 * x1 * x7-6.0 * x3 * x5);
		z12 = -6.0 *  (a1 * a6 + a3 * a5) + emsq *
			(-24.0 * (x2 * x7 + x1 * x8) - 6.0 * (x3 * x6 + x4 * x5));
		z13 = -6.0 * a3 * a6 + emsq * (-24.0 * x2 * x8 - 6.0 * x4 * x6);
		z21 =  6.0 * a2 * a5 + emsq * (24.0 * x1 * x5 - 6.0 * x3 * x7);
		z22 =  6.0 *  (a4 * a5 + a2 * a6) + emsq *
			(24.0 * (x2 * x5 + x1 * x6) - 6.0 * (x4 * x7 + x3 * x8));
		z23 =  6.0 * a4 * a6 + emsq * (24.0 * x2 * x6 - 6.0 * x4 * x8);
		z1  = z1 + z1 + betasq * z31;
		z2  = z2 + z2 + betasq * z32;
		z3  = z3 + z3 + betasq * z33;
		s3  = cc * xnoi;
		s2  = -0.5 * s3 / rtemsq;
		s4  = s3 * rtemsq;
		s1  = -15.0 * em * s4;
		s5  = x1 * x3 + x2 * x4;
		s6  = x2 * x3 + x1 * x4;
		s7  = x2 * x4 - x1 * x3;

		/* ----------------------- do lunar terms ------------------- */
		if (lsflg == 1)
		{
			ss1   = s1;
			ss2   = s2;
			ss3   = s3;
			ss4   = s4;
			ss5   = s5;
			ss6   = s6;
			ss7   = s7;
			sz1   = z1;
			sz2   = z2;
			sz3   = z3;
			sz11  = z11;
			sz12  = z12;
			sz13  = z13;
			sz21  = z21;
			sz22  = z22;
			sz23  = z23;
			sz31  = z31;
			sz32  = z32;
			sz33  = z33;
			zcosg = zcosgl;
			zsing = zsingl;
			zcosi = zcosil;
			zsini = zsinil;
			zcosh = zcoshl * cnodm + zsinhl * snodm;
			zsinh = snodm * zcoshl - cnodm * zsinhl;
			cc    = c1l;
		}
	}

	zmol = fmod(4.7199672 + 0.22997150  * day - gam, twopi);
	zmos = fmod(6.2565837 + 0.017201977 * day, twopi);

	/* ------------------------ do solar terms ---------------------- */
	se2  =   2.0 * ss1 * ss6;
	se3  =   2.0 * ss1 * ss7;
	si2  =   2.0 * ss2 * sz12;
	si3  =   2.0 * ss2 * (sz13 - sz11);
	sl2  =  -2.0 * ss3 * sz2;
	sl3  =  -2.0 * ss3 * (sz3 - sz1);
	sl4  =  -2.0 * ss3 * (-21.0 - 9.0 * emsq) * zes;
	sgh2 =   2.0 * ss4 * sz32;
	sgh3 =   2.0 * ss4 * (sz33 - sz31);
	sgh4 = -18.0 * ss4 * zes;
	sh2  =  -2.0 * ss2 * sz22;
	sh3  =  -2.0 * ss2 * (sz23 - sz21);

	/* ------------------------ do lunar terms ---------------------- */
	ee2  =   2.0 * s1 * s6;
	e3   =   2.0 * s1 * s7;
	xi2  =   2.0 * s2 * z12;
	xi3  =   2.0 * s2 * (z13 - z11);
	xl2  =  -2.0 * s3 * z2;
	xl3  =  -2.0 * s3 * (z3 - z1);
	xl4  =  -2.0 * s3 * (-21.0 - 9.0 * emsq) * zel;
	xgh2 =   2.0 * s4 * z32;
	xgh3 =   2.0 * s4 * (z33 - z31);
	xgh4 = -18.0 * s4 * zel;
	xh2  =  -2.0 * s2 * z22;
	xh3  =  -2.0 * s2 * (z23 - z21);

	//#include "debug2.cpp"
}  // end dscom

//static void initl
//	(
//	int satn,      gravconsttype whichconst,
//	double ecco,   double epoch,  double inclo,   double& no,
//	char& method,
//	double& ainv,  double& ao,    double& con41,  double& con42, double& cosio,
//	double& cosio2,double& eccsq, double& omeosq, double& posq,
//	double& rp,    double& rteosq,double& sinio , double& gsto
//	)
//{
//	/* --------------------- local variables ------------------------ */
//	double ak, d1, del, adel, po, x2o3, j2, xke,
//		tumin, mu, radiusearthkm, j3, j4, j3oj2;
//
//	// sgp4fix use old way of finding gst
//	int ids70;
//	double ts70, ds70, tfrac, c1, thgr70, fk5r, c1p2p, thgr, thgro;
//	const double twopi = 2.0 * PI;
//
//	/* ----------------------- earth constants ---------------------- */
//	// sgp4fix identify constants and allow alternate values
//	//getgravconst( whichconst, tumin, mu, radiusearthkm, xke, j2, j3, j4, j3oj2 );
//	x2o3   = 2.0 / 3.0;
//
//	/* ------------- calculate auxillary epoch quantities ---------- */
//	eccsq  = ecco * ecco;
//	omeosq = 1.0 - eccsq;
//	rteosq = sqrt(omeosq);
//	cosio  = cos(inclo);
//	cosio2 = cosio * cosio;
//
//	/* ------------------ un-kozai the mean motion ----------------- */
//	ak    = pow(gravity_constants.xke / no, x2o3);
//	d1    = 0.75 * gravity_constants.j2 * (3.0 * cosio2 - 1.0) / (rteosq * omeosq);
//	del   = d1 / (ak * ak);
//	adel  = ak * (1.0 - del * del - del *
//		(1.0 / 3.0 + 134.0 * del * del / 81.0));
//	del   = d1/(adel * adel);
//	no    = no / (1.0 + del);
//
//	ao    = pow(xke / no, x2o3);
//	sinio = sin(inclo);
//	po    = ao * omeosq;
//	con42 = 1.0 - 5.0 * cosio2;
//	con41 = -con42-cosio2-cosio2;
//	ainv  = 1.0 / ao;
//	posq  = po * po;
//	rp    = ao * (1.0 - ecco);
//	method = 'n';
//
//	// sgp4fix modern approach to finding sidereal timew
//	// gsto = gstime(epoch + 2433281.5);
//
//	//// sgp4fix use old way of finding gst
//	//// count integer number of days from 0 jan 1970
//	//ts70  = epoch - 7305.0;
//	//ids70 = floor(ts70 + 1.0e-8);
//	//ds70  = ids70;
//	//tfrac = ts70 - ds70;
//	//// find greenwich location at epoch
//	//c1    = 1.72027916940703639e-2;
//	//thgr70= 1.7321343856509374;
//	//fk5r  = 5.07551419432269442e-15;
//	//c1p2p = c1 + twopi;
//	//gsto  = fmod( thgr70 + c1*ds70 + c1p2p*tfrac + ts70*ts70*fk5r, twopi);
//	//if ( gsto < 0.0 )
//	//	gsto = gsto + twopi;
//
//	//#include "debug5.cpp"
//}  // end initl



/* -----------------------------------------------------------------------------
*
*                           procedure dpper
*
*  this procedure provides deep space long period periodic contributions
*    to the mean elements.  by design, these periodics are zero at epoch.
*    this used to be dscom which included initialization, but it's really a
*    recurring function.
*
*  author        : david vallado                  719-573-2600   28 jun 2005
*
*  inputs        :
*    e3          -
*    ee2         -
*    peo         -
*    pgho        -
*    pho         -
*    pinco       -
*    plo         -
*    se2 , se3 , sgh2, sgh3, sgh4, sh2, sh3, si2, si3, sl2, sl3, sl4 -
*    t           -
*    xh2, xh3, xi2, xi3, xl2, xl3, xl4 -
*    zmol        -
*    zmos        -
*    ep          - eccentricity                           0.0 - 1.0
*    inclo       - inclination - needed for lyddane modification
*    nodep       - right ascension of ascending node
*    argpp       - argument of perigee
*    mp          - mean anomaly
*
*  outputs       :
*    ep          - eccentricity                           0.0 - 1.0
*    inclp       - inclination
*    nodep        - right ascension of ascending node
*    argpp       - argument of perigee
*    mp          - mean anomaly
*
*  locals        :
*    alfdp       -
*    betdp       -
*    cosip  , sinip  , cosop  , sinop  ,
*    dalf        -
*    dbet        -
*    dls         -
*    f2, f3      -
*    pe          -
*    pgh         -
*    ph          -
*    pinc        -
*    pl          -
*    sel   , ses   , sghl  , sghs  , shl   , shs   , sil   , sinzf , sis   ,
*    sll   , sls
*    xls         -
*    xnoh        -
*    zf          -
*    zm          -
*
*  coupling      :
*    none.
*
*  references    :
*    hoots, roehrich, norad spacetrack report #3 1980
*    hoots, norad spacetrack report #6 1986
*    hoots, schumacher and glover 2004
*    vallado, crawford, hujsak, kelso  2006
----------------------------------------------------------------------------*/


__device__ static void dpper
	(
	double e3,     double ee2,    double peo,     double pgho,   double pho,
	double pinco,  double plo,    double se2,     double se3,    double sgh2,
	double sgh3,   double sgh4,   double sh2,     double sh3,    double si2,
	double si3,    double sl2,    double sl3,     double sl4,    double t,
	double xgh2,   double xgh3,   double xgh4,    double xh2,    double xh3,
	double xi2,    double xi3,    double xl2,     double xl3,    double xl4,
	double zmol,   double zmos,   double inclo,
	char init,
	double& ep,    double& inclp, double& nodep,  double& argpp, double& mp
	)
{
	/* --------------------- local variables ------------------------ */
	const double twopi = 2.0 * HIP_PI;
	double alfdp, betdp, cosip, cosop, dalf, dbet, dls,
		f2,    f3,    pe,    pgh,   ph,   pinc, pl ,
		sel,   ses,   sghl,  sghs,  shll, shs,  sil,
		sinip, sinop, sinzf, sis,   sll,  sls,  xls,
		xnoh,  zf,    zm,    zel,   zes,  znl,  zns;

	/* ---------------------- constants ----------------------------- */
	zns   = 1.19459e-5;
	zes   = 0.01675;
	znl   = 1.5835218e-4;
	zel   = 0.05490;

	/* --------------- calculate time varying periodics ----------- */
	zm    = zmos + zns * t;
	// be sure that the initial call has time set to zero
	if (init == 'y')
		zm = zmos;
	zf    = zm + 2.0 * zes * sin(zm);
	sinzf = sin(zf);
	f2    =  0.5 * sinzf * sinzf - 0.25;
	f3    = -0.5 * sinzf * cos(zf);
	ses   = se2* f2 + se3 * f3;
	sis   = si2 * f2 + si3 * f3;
	sls   = sl2 * f2 + sl3 * f3 + sl4 * sinzf;
	sghs  = sgh2 * f2 + sgh3 * f3 + sgh4 * sinzf;
	shs   = sh2 * f2 + sh3 * f3;
	zm    = zmol + znl * t;
	if (init == 'y')
		zm = zmol;
	zf    = zm + 2.0 * zel * sin(zm);
	sinzf = sin(zf);
	f2    =  0.5 * sinzf * sinzf - 0.25;
	f3    = -0.5 * sinzf * cos(zf);
	sel   = ee2 * f2 + e3 * f3;
	sil   = xi2 * f2 + xi3 * f3;
	sll   = xl2 * f2 + xl3 * f3 + xl4 * sinzf;
	sghl  = xgh2 * f2 + xgh3 * f3 + xgh4 * sinzf;
	shll  = xh2 * f2 + xh3 * f3;
	pe    = ses + sel;
	pinc  = sis + sil;
	pl    = sls + sll;
	pgh   = sghs + sghl;
	ph    = shs + shll;

	if (init == 'n')
	{
		pe    = pe - peo;
		pinc  = pinc - pinco;
		pl    = pl - plo;
		pgh   = pgh - pgho;
		ph    = ph - pho;
		inclp = inclp + pinc;
		ep    = ep + pe;
		sinip = sin(inclp);
		cosip = cos(inclp);

		/* ----------------- apply periodics directly ------------ */
		//  sgp4fix for lyddane choice
		//  strn3 used original inclination - this is technically feasible
		//  gsfc used perturbed inclination - also technically feasible
		//  probably best to readjust the 0.2 limit value and limit discontinuity
		//  0.2 rad = 11.45916 deg
		//  use next line for original strn3 approach and original inclination
		//  if (inclo >= 0.2)
		//  use next line for gsfc version and perturbed inclination
		if (inclp >= 0.2)
		{
			ph     = ph / sinip;
			pgh    = pgh - cosip * ph;
			argpp  = argpp + pgh;
			nodep  = nodep + ph;
			mp     = mp + pl;
		}
		else
		{
			/* ---- apply periodics with lyddane modification ---- */
			sinop  = sin(nodep);
			cosop  = cos(nodep);
			alfdp  = sinip * sinop;
			betdp  = sinip * cosop;
			dalf   =  ph * cosop + pinc * cosip * sinop;
			dbet   = -ph * sinop + pinc * cosip * cosop;
			alfdp  = alfdp + dalf;
			betdp  = betdp + dbet;
			nodep  = fmod(nodep, twopi);
			//  sgp4fix for afspc written intrinsic functions
			// nodep used without a trigonometric function ahead
			if (nodep < 0.0)
				nodep = nodep + twopi;
			xls    = mp + argpp + cosip * nodep;
			dls    = pl + pgh - pinc * nodep * sinip;
			xls    = xls + dls;
			xnoh   = nodep;
			nodep  = atan2(alfdp, betdp);
			//  sgp4fix for afspc written intrinsic functions
			// nodep used without a trigonometric function ahead
			if (nodep < 0.0)
				nodep = nodep + twopi;
			if (fabs(xnoh - nodep) > HIP_PI)
				if (nodep < xnoh)
					nodep = nodep + twopi;
				else
					nodep = nodep - twopi;
			mp    = mp + pl;
			argpp = xls - mp - cosip * nodep;
		}
	}   // if init == 'n'

	//#include "debug1.cpp"
}  // end dpper


/*-----------------------------------------------------------------------------
*
*                           procedure dsinit
*
*  this procedure provides deep space contributions to mean motion dot due
*    to geopotential resonance with half day and one day orbits.
*
*  author        : david vallado                  719-573-2600   28 jun 2005
*
*  inputs        :
*    cosim, sinim-
*    emsq        - eccentricity squared
*    argpo       - argument of perigee
*    s1, s2, s3, s4, s5      -
*    ss1, ss2, ss3, ss4, ss5 -
*    sz1, sz3, sz11, sz13, sz21, sz23, sz31, sz33 -
*    t           - time
*    tc          -
*    gsto        - greenwich sidereal time                   rad
*    mo          - mean anomaly
*    mdot        - mean anomaly dot (rate)
*    no          - mean motion
*    nodeo       - right ascension of ascending node
*    nodedot     - right ascension of ascending node dot (rate)
*    xpidot      -
*    z1, z3, z11, z13, z21, z23, z31, z33 -
*    eccm        - eccentricity
*    argpm       - argument of perigee
*    inclm       - inclination
*    mm          - mean anomaly
*    xn          - mean motion
*    nodem       - right ascension of ascending node
*
*  outputs       :
*    em          - eccentricity
*    argpm       - argument of perigee
*    inclm       - inclination
*    mm          - mean anomaly
*    nm          - mean motion
*    nodem       - right ascension of ascending node
*    irez        - flag for resonance           0-none, 1-one day, 2-half day
*    atime       -
*    d2201, d2211, d3210, d3222, d4410, d4422, d5220, d5232, d5421, d5433    -
*    dedt        -
*    didt        -
*    dmdt        -
*    dndt        -
*    dnodt       -
*    domdt       -
*    del1, del2, del3        -
*    ses  , sghl , sghs , sgs  , shl  , shs  , sis  , sls
*    theta       -
*    xfact       -
*    xlamo       -
*    xli         -
*    xni
*
*  locals        :
*    ainv2       -
*    aonv        -
*    cosisq      -
*    eoc         -
*    f220, f221, f311, f321, f322, f330, f441, f442, f522, f523, f542, f543  -
*    g200, g201, g211, g300, g310, g322, g410, g422, g520, g521, g532, g533  -
*    sini2       -
*    temp        -
*    temp1       -
*    theta       -
*    xno2        -
*
*  coupling      :
*    getgravconst
*
*  references    :
*    hoots, roehrich, norad spacetrack report #3 1980
*    hoots, norad spacetrack report #6 1986
*    hoots, schumacher and glover 2004
*    vallado, crawford, hujsak, kelso  2006
----------------------------------------------------------------------------*/

__device__ static void dsinit
	(
	double cosim,  double emsq,   double argpo,   double s1,     double s2,
	double s3,     double s4,     double s5,      double sinim,  double ss1,
	double ss2,    double ss3,    double ss4,     double ss5,    double sz1,
	double sz3,    double sz11,   double sz13,    double sz21,   double sz23,
	double sz31,   double sz33,   double t,       double tc,     double gsto,
	double mo,     double mdot,   double no,      double nodeo,  double nodedot,
	double xpidot, double z1,     double z3,      double z11,    double z13,
	double z21,    double z23,    double z31,     double z33,    double ecco,
	double eccsq,  double& em,    double& argpm,  double& inclm, double& mm,
	double& nm,    double& nodem,
	int& irez,
	double& atime, double& d2201, double& d2211,  double& d3210, double& d3222,
	double& d4410, double& d4422, double& d5220,  double& d5232, double& d5421,
	double& d5433, double& dedt,  double& didt,   double& dmdt,  double& dndt,
	double& dnodt, double& domdt, double& del1,   double& del2,  double& del3,
	double& xfact, double& xlamo, double& xli,    double& xni
	)
{
	/* --------------------- local variables ------------------------ */
	const double twopi = 2.0 * HIP_PI;

	double ainv2 , aonv=0.0, cosisq, eoc, f220 , f221  , f311  ,
		f321  , f322  , f330  , f441  , f442  , f522  , f523  ,
		f542  , f543  , g200  , g201  , g211  , g300  , g310  ,
		g322  , g410  , g422  , g520  , g521  , g532  , g533  ,
		ses   , sgs   , sghl  , sghs  , shs   , shll  , sis   ,
		sini2 , sls   , temp  , temp1 , theta , xno2  , q22   ,
		q31   , q33   , root22, root44, root54, rptim , root32,
		root52, x2o3  , /*xke   ,*/ znl   , emo   , zns   , emsqo
		//,tumin, mu, radiusearthkm, j2, j3, j4, j3oj2
		;

	q22    = 1.7891679e-6;
	q31    = 2.1460748e-6;
	q33    = 2.2123015e-7;
	root22 = 1.7891679e-6;
	root44 = 7.3636953e-9;
	root54 = 2.1765803e-9;
	rptim  = 4.37526908801129966e-3; // this equates to 7.29211514668855e-5 rad/sec
	root32 = 3.7393792e-7;
	root52 = 1.1428639e-7;
	x2o3   = 2.0 / 3.0;
	znl    = 1.5835218e-4;
	zns    = 1.19459e-5;

	// sgp4fix identify constants and allow alternate values
	//getgravconst( whichconst, tumin, mu, radiusearthkm, xke, j2, j3, j4, j3oj2 );

	/* -------------------- deep space initialization ------------ */
	irez = 0;
	if ((nm < 0.0052359877) && (nm > 0.0034906585))
		irez = 1;
	if ((nm >= 8.26e-3) && (nm <= 9.24e-3) && (em >= 0.5))
		irez = 2;

	/* ------------------------ do solar terms ------------------- */
	ses  =  ss1 * zns * ss5;
	sis  =  ss2 * zns * (sz11 + sz13);
	sls  = -zns * ss3 * (sz1 + sz3 - 14.0 - 6.0 * emsq);
	sghs =  ss4 * zns * (sz31 + sz33 - 6.0);
	shs  = -zns * ss2 * (sz21 + sz23);
	// sgp4fix for 180 deg incl
	if ((inclm < 5.2359877e-2) || (inclm > HIP_PI - 5.2359877e-2))
		shs = 0.0;
	if (sinim != 0.0)
		shs = shs / sinim;
	sgs  = sghs - cosim * shs;

	/* ------------------------- do lunar terms ------------------ */
	dedt = ses + s1 * znl * s5;
	didt = sis + s2 * znl * (z11 + z13);
	dmdt = sls - znl * s3 * (z1 + z3 - 14.0 - 6.0 * emsq);
	sghl = s4 * znl * (z31 + z33 - 6.0);
	shll = -znl * s2 * (z21 + z23);
	// sgp4fix for 180 deg incl
	if ((inclm < 5.2359877e-2) || (inclm > HIP_PI - 5.2359877e-2))
		shll = 0.0;
	domdt = sgs + sghl;
	dnodt = shs;
	if (sinim != 0.0)
	{
		domdt = domdt - cosim / sinim * shll;
		dnodt = dnodt + shll / sinim;
	}

	/* ----------- calculate deep space resonance effects -------- */
	dndt   = 0.0;
	theta  = fmod(gsto + tc * rptim, twopi);
	em     = em + dedt * t;
	inclm  = inclm + didt * t;
	argpm  = argpm + domdt * t;
	nodem  = nodem + dnodt * t;
	mm     = mm + dmdt * t;
	//   sgp4fix for negative inclinations
	//   the following if statement should be commented out
	//if (inclm < 0.0)
	//  {
	//    inclm  = -inclm;
	//    argpm  = argpm - pi;
	//    nodem = nodem + pi;
	//  }

	/* -------------- initialize the resonance terms ------------- */
	if (irez != 0)
	{
		aonv = pow(nm / gravity_constants.xke, x2o3);

		/* ---------- geopotential resonance for 12 hour orbits ------ */
		if (irez == 2)
		{
			cosisq = cosim * cosim;
			emo    = em;
			em     = ecco;
			emsqo  = emsq;
			emsq   = eccsq;
			eoc    = em * emsq;
			g201   = -0.306 - (em - 0.64) * 0.440;

			if (em <= 0.65)
			{
				g211 =    3.616  -  13.2470 * em +  16.2900 * emsq;
				g310 =  -19.302  + 117.3900 * em - 228.4190 * emsq +  156.5910 * eoc;
				g322 =  -18.9068 + 109.7927 * em - 214.6334 * emsq +  146.5816 * eoc;
				g410 =  -41.122  + 242.6940 * em - 471.0940 * emsq +  313.9530 * eoc;
				g422 = -146.407  + 841.8800 * em - 1629.014 * emsq + 1083.4350 * eoc;
				g520 = -532.114  + 3017.977 * em - 5740.032 * emsq + 3708.2760 * eoc;
			}
			else
			{
				g211 =   -72.099 +   331.819 * em -   508.738 * emsq +   266.724 * eoc;
				g310 =  -346.844 +  1582.851 * em -  2415.925 * emsq +  1246.113 * eoc;
				g322 =  -342.585 +  1554.908 * em -  2366.899 * emsq +  1215.972 * eoc;
				g410 = -1052.797 +  4758.686 * em -  7193.992 * emsq +  3651.957 * eoc;
				g422 = -3581.690 + 16178.110 * em - 24462.770 * emsq + 12422.520 * eoc;
				if (em > 0.715)
					g520 =-5149.66 + 29936.92 * em - 54087.36 * emsq + 31324.56 * eoc;
				else
					g520 = 1464.74 -  4664.75 * em +  3763.64 * emsq;
			}
			if (em < 0.7)
			{
				g533 = -919.22770 + 4988.6100 * em - 9064.7700 * emsq + 5542.21  * eoc;
				g521 = -822.71072 + 4568.6173 * em - 8491.4146 * emsq + 5337.524 * eoc;
				g532 = -853.66600 + 4690.2500 * em - 8624.7700 * emsq + 5341.4  * eoc;
			}
			else
			{
				g533 =-37995.780 + 161616.52 * em - 229838.20 * emsq + 109377.94 * eoc;
				g521 =-51752.104 + 218913.95 * em - 309468.16 * emsq + 146349.42 * eoc;
				g532 =-40023.880 + 170470.89 * em - 242699.48 * emsq + 115605.82 * eoc;
			}

			sini2=  sinim * sinim;
			f220 =  0.75 * (1.0 + 2.0 * cosim+cosisq);
			f221 =  1.5 * sini2;
			f321 =  1.875 * sinim  *  (1.0 - 2.0 * cosim - 3.0 * cosisq);
			f322 = -1.875 * sinim  *  (1.0 + 2.0 * cosim - 3.0 * cosisq);
			f441 = 35.0 * sini2 * f220;
			f442 = 39.3750 * sini2 * sini2;
			f522 =  9.84375 * sinim * (sini2 * (1.0 - 2.0 * cosim- 5.0 * cosisq) +
				0.33333333 * (-2.0 + 4.0 * cosim + 6.0 * cosisq) );
			f523 = sinim * (4.92187512 * sini2 * (-2.0 - 4.0 * cosim +
				10.0 * cosisq) + 6.56250012 * (1.0+2.0 * cosim - 3.0 * cosisq));
			f542 = 29.53125 * sinim * (2.0 - 8.0 * cosim+cosisq *
				(-12.0 + 8.0 * cosim + 10.0 * cosisq));
			f543 = 29.53125 * sinim * (-2.0 - 8.0 * cosim+cosisq *
				(12.0 + 8.0 * cosim - 10.0 * cosisq));
			xno2  =  nm * nm;
			ainv2 =  aonv * aonv;
			temp1 =  3.0 * xno2 * ainv2;
			temp  =  temp1 * root22;
			d2201 =  temp * f220 * g201;
			d2211 =  temp * f221 * g211;
			temp1 =  temp1 * aonv;
			temp  =  temp1 * root32;
			d3210 =  temp * f321 * g310;
			d3222 =  temp * f322 * g322;
			temp1 =  temp1 * aonv;
			temp  =  2.0 * temp1 * root44;
			d4410 =  temp * f441 * g410;
			d4422 =  temp * f442 * g422;
			temp1 =  temp1 * aonv;
			temp  =  temp1 * root52;
			d5220 =  temp * f522 * g520;
			d5232 =  temp * f523 * g532;
			temp  =  2.0 * temp1 * root54;
			d5421 =  temp * f542 * g521;
			d5433 =  temp * f543 * g533;
			xlamo =  fmod(mo + nodeo + nodeo-theta - theta, twopi);
			xfact =  mdot + dmdt + 2.0 * (nodedot + dnodt - rptim) - no;
			em    = emo;
			emsq  = emsqo;
		}

		/* ---------------- synchronous resonance terms -------------- */
		if (irez == 1)
		{
			g200  = 1.0 + emsq * (-2.5 + 0.8125 * emsq);
			g310  = 1.0 + 2.0 * emsq;
			g300  = 1.0 + emsq * (-6.0 + 6.60937 * emsq);
			f220  = 0.75 * (1.0 + cosim) * (1.0 + cosim);
			f311  = 0.9375 * sinim * sinim * (1.0 + 3.0 * cosim) - 0.75 * (1.0 + cosim);
			f330  = 1.0 + cosim;
			f330  = 1.875 * f330 * f330 * f330;
			del1  = 3.0 * nm * nm * aonv * aonv;
			del2  = 2.0 * del1 * f220 * g200 * q22;
			del3  = 3.0 * del1 * f330 * g300 * q33 * aonv;
			del1  = del1 * f311 * g310 * q31 * aonv;
			xlamo = fmod(mo + nodeo + argpo - theta, twopi);
			xfact = mdot + xpidot - rptim + dmdt + domdt + dnodt - no;
		}

		/* ------------ for sgp4, initialize the integrator ---------- */
		xli   = xlamo;
		xni   = no;
		atime = 0.0;
		nm    = no + dndt;
	}

	//#include "debug3.cpp"
}  // end dsinit