#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2011 NVIDIA Corporation.  All rights reserved.
 *
 * NVIDIA Corporation and its licensors retain all intellectual property and 
 * proprietary rights in and to this software and related documentation. 
 * Any use, reproduction, disclosure, or distribution of this software 
 * and related documentation without an express license agreement from
 * NVIDIA Corporation is strictly prohibited.
 *
 * Please refer to the applicable NVIDIA end user license agreement (EULA) 
 * associated with this source code for terms and conditions that govern 
 * your use of this NVIDIA software.
 * 
 */

#include "common.h"
#include "commonCUDA.cu"
#include "satelliterecord.h"

__global__ void sgp4init
	(gravconsttype whichconst,       const int satn,     const double epoch,
	const double xbstar,  const double xecco, const double xargpo,
	const double xinclo,  const double xmo,   const double xno,
	const double xnodeo,  elsetrec& satrec
	)
{
	/* --------------------- local variables ------------------------ */
	double ao, ainv,   con42, cosio, sinio, cosio2, eccsq,
		omeosq, posq,   rp,     rteosq,
		cnodm , snodm , cosim , sinim , cosomm, sinomm, cc1sq ,
		cc2   , cc3   , coef  , coef1 , cosio4, day   , dndt  ,
		em    , emsq  , eeta  , etasq , gam   , argpm , nodem ,
		inclm , mm    , nm    , perige, pinvsq, psisq , qzms24,
		rtemsq, s1    , s2    , s3    , s4    , s5    , s6    ,
		s7    , sfour , ss1   , ss2   , ss3   , ss4   , ss5   ,
		ss6   , ss7   , sz1   , sz2   , sz3   , sz11  , sz12  ,
		sz13  , sz21  , sz22  , sz23  , sz31  , sz32  , sz33  ,
		tc    , temp  , temp1 , temp2 , temp3 , tsi   , xpidot,
		xhdot1, z1    , z2    , z3    , z11   , z12   , z13   ,
		z21   , z22   , z23   , z31   , z32   , z33,
		qzms2t, ss, j2, j3oj2, j4, x2o3, r[3], v[3],
		tumin, mu, radiusearthkm, xke, j3;

	/* ------------------------ initialization --------------------- */
	// sgp4fix divisor for divide by zero check on inclination
	const double temp4    =   1.0 + cos(pi-1.0e-9);

	/* ----------- set all near earth variables to zero ------------ */
	satrec.isimp   = 0;   satrec.method = 'n'; satrec.aycof    = 0.0;
	satrec.con41   = 0.0; satrec.cc1    = 0.0; satrec.cc4      = 0.0;
	satrec.cc5     = 0.0; satrec.d2     = 0.0; satrec.d3       = 0.0;
	satrec.d4      = 0.0; satrec.delmo  = 0.0; satrec.eta      = 0.0;
	satrec.argpdot = 0.0; satrec.omgcof = 0.0; satrec.sinmao   = 0.0;
	satrec.t       = 0.0; satrec.t2cof  = 0.0; satrec.t3cof    = 0.0;
	satrec.t4cof   = 0.0; satrec.t5cof  = 0.0; satrec.x1mth2   = 0.0;
	satrec.x7thm1  = 0.0; satrec.mdot   = 0.0; satrec.nodedot  = 0.0;
	satrec.xlcof   = 0.0; satrec.xmcof  = 0.0; satrec.nodecf   = 0.0;

	/* ----------- set all deep space variables to zero ------------ */
	satrec.irez  = 0;   satrec.d2201 = 0.0; satrec.d2211 = 0.0;
	satrec.d3210 = 0.0; satrec.d3222 = 0.0; satrec.d4410 = 0.0;
	satrec.d4422 = 0.0; satrec.d5220 = 0.0; satrec.d5232 = 0.0;
	satrec.d5421 = 0.0; satrec.d5433 = 0.0; satrec.dedt  = 0.0;
	satrec.del1  = 0.0; satrec.del2  = 0.0; satrec.del3  = 0.0;
	satrec.didt  = 0.0; satrec.dmdt  = 0.0; satrec.dnodt = 0.0;
	satrec.domdt = 0.0; satrec.e3    = 0.0; satrec.ee2   = 0.0;
	satrec.peo   = 0.0; satrec.pgho  = 0.0; satrec.pho   = 0.0;
	satrec.pinco = 0.0; satrec.plo   = 0.0; satrec.se2   = 0.0;
	satrec.se3   = 0.0; satrec.sgh2  = 0.0; satrec.sgh3  = 0.0;
	satrec.sgh4  = 0.0; satrec.sh2   = 0.0; satrec.sh3   = 0.0;
	satrec.si2   = 0.0; satrec.si3   = 0.0; satrec.sl2   = 0.0;
	satrec.sl3   = 0.0; satrec.sl4   = 0.0; satrec.gsto  = 0.0;
	satrec.xfact = 0.0; satrec.xgh2  = 0.0; satrec.xgh3  = 0.0;
	satrec.xgh4  = 0.0; satrec.xh2   = 0.0; satrec.xh3   = 0.0;
	satrec.xi2   = 0.0; satrec.xi3   = 0.0; satrec.xl2   = 0.0;
	satrec.xl3   = 0.0; satrec.xl4   = 0.0; satrec.xlamo = 0.0;
	satrec.zmol  = 0.0; satrec.zmos  = 0.0; satrec.atime = 0.0;
	satrec.xli   = 0.0; satrec.xni   = 0.0;

	// sgp4fix - note the following variables are also passed directly via satrec.
	// it is possible to streamline the sgp4init call by deleting the "x"
	// variables, but the user would need to set the satrec.* values first. we
	// include the additional assignments in case twoline2rv is not used.
	satrec.bstar   = xbstar;
	satrec.ecco    = xecco;
	satrec.argpo   = xargpo;
	satrec.inclo   = xinclo;
	satrec.mo	    = xmo;
	satrec.no	    = xno;
	satrec.nodeo   = xnodeo;

	/* ------------------------ earth constants ----------------------- */
	// sgp4fix identify constants and allow alternate values
	getgravconst( whichconst, tumin, mu, radiusearthkm, xke, j2, j3, j4, j3oj2 );
	ss     = 78.0 / radiusearthkm + 1.0;
	qzms2t = pow(((120.0 - 78.0) / radiusearthkm), 4);
	x2o3   =  2.0 / 3.0;

	satrec.init = 'y';
	satrec.t	 = 0.0;

	initl
		(
		satn, whichconst, satrec.ecco, epoch, satrec.inclo, satrec.no, satrec.method,
		ainv, ao, satrec.con41, con42, cosio, cosio2, eccsq, omeosq,
		posq, rp, rteosq, sinio, satrec.gsto
		);
	satrec.error = 0;

	if (rp < 1.0)
	{
		//         printf("# *** satn%d epoch elts sub-orbital ***\n", satn);
		satrec.error = 5;
	}

	if ((omeosq >= 0.0 ) || ( satrec.no >= 0.0))
	{
		satrec.isimp = 0;
		if (rp < (220.0 / radiusearthkm + 1.0))
			satrec.isimp = 1;
		sfour  = ss;
		qzms24 = qzms2t;
		perige = (rp - 1.0) * radiusearthkm;

		/* - for perigees below 156 km, s and qoms2t are altered - */
		if (perige < 156.0)
		{
			sfour = perige - 78.0;
			if (perige < 98.0)
				sfour = 20.0;
			qzms24 = pow(((120.0 - sfour) / radiusearthkm), 4.0);
			sfour  = sfour / radiusearthkm + 1.0;
		}
		pinvsq = 1.0 / posq;

		tsi  = 1.0 / (ao - sfour);
		satrec.eta  = ao * satrec.ecco * tsi;
		etasq = satrec.eta * satrec.eta;
		eeta  = satrec.ecco * satrec.eta;
		psisq = fabs(1.0 - etasq);
		coef  = qzms24 * pow(tsi, 4.0);
		coef1 = coef / pow(psisq, 3.5);
		cc2   = coef1 * satrec.no * (ao * (1.0 + 1.5 * etasq + eeta *
			(4.0 + etasq)) + 0.375 * j2 * tsi / psisq * satrec.con41 *
			(8.0 + 3.0 * etasq * (8.0 + etasq)));
		satrec.cc1   = satrec.bstar * cc2;
		cc3   = 0.0;
		if (satrec.ecco > 1.0e-4)
			cc3 = -2.0 * coef * tsi * j3oj2 * satrec.no * sinio / satrec.ecco;
		satrec.x1mth2 = 1.0 - cosio2;
		satrec.cc4    = 2.0* satrec.no * coef1 * ao * omeosq *
			(satrec.eta * (2.0 + 0.5 * etasq) + satrec.ecco *
			(0.5 + 2.0 * etasq) - j2 * tsi / (ao * psisq) *
			(-3.0 * satrec.con41 * (1.0 - 2.0 * eeta + etasq *
			(1.5 - 0.5 * eeta)) + 0.75 * satrec.x1mth2 *
			(2.0 * etasq - eeta * (1.0 + etasq)) * cos(2.0 * satrec.argpo)));
		satrec.cc5 = 2.0 * coef1 * ao * omeosq * (1.0 + 2.75 *
			(etasq + eeta) + eeta * etasq);
		cosio4 = cosio2 * cosio2;
		//temp1  = 1.5 * j2 * pinvsq * satrec.no;
		temp2  = 0.5 * 1.5 * j2 * pinvsq * satrec.no * j2 * pinvsq;
		temp3  = -0.46875 * j4 * pinvsq * pinvsq * satrec.no;
		satrec.mdot     = satrec.no + 0.5 * 1.5 * j2 * pinvsq * satrec.no * rteosq * satrec.con41 + 0.0625 *
			temp2 * rteosq * (13.0 - 78.0 * cosio2 + 137.0 * cosio4);
		satrec.argpdot  = -0.5 * 1.5 * j2 * pinvsq * satrec.no * con42 + 0.0625 * temp2 *
			(7.0 - 114.0 * cosio2 + 395.0 * cosio4) +
			temp3 * (3.0 - 36.0 * cosio2 + 49.0 * cosio4);
		xhdot1            = -1.5 * j2 * pinvsq * satrec.no * cosio;
		satrec.nodedot = xhdot1 + (0.5 * temp2 * (4.0 - 19.0 * cosio2) +
			2.0 * temp3 * (3.0 - 7.0 * cosio2)) * cosio;
		xpidot            =  satrec.argpdot+ satrec.nodedot;
		satrec.omgcof   = satrec.bstar * cc3 * cos(satrec.argpo);
		satrec.xmcof    = 0.0;
		if (satrec.ecco > 1.0e-4)
			satrec.xmcof = -x2o3 * coef * satrec.bstar / eeta;
		satrec.nodecf = 3.5 * omeosq * xhdot1 * satrec.cc1;
		satrec.t2cof   = 1.5 * satrec.cc1;
		// sgp4fix for divide by zero with xinco = 180 deg
		if (fabs(cosio+1.0) > 1.5e-12)
			satrec.xlcof = -0.25 * j3oj2 * sinio * (3.0 + 5.0 * cosio) / (1.0 + cosio);
		else
			satrec.xlcof = -0.25 * j3oj2 * sinio * (3.0 + 5.0 * cosio) / temp4;
		satrec.aycof   = -0.5 * j3oj2 * sinio;
		satrec.delmo   = pow((1.0 + satrec.eta * cos(satrec.mo)), 3);
		satrec.sinmao  = sin(satrec.mo);
		satrec.x7thm1  = 7.0 * cosio2 - 1.0;

		/* --------------- deep space initialization ------------- */
		if ((2*pi / satrec.no) >= 225.0)
		{
			satrec.method = 'd';
			satrec.isimp  = 1;
			tc    =  0.0;
			inclm = satrec.inclo;

			dscom
				(
				epoch, satrec.ecco, satrec.argpo, tc, satrec.inclo, satrec.nodeo,
				satrec.no, snodm, cnodm,  sinim, cosim,sinomm,     cosomm,
				day, satrec.e3, satrec.ee2, em,         emsq, gam,
				satrec.peo,  satrec.pgho,   satrec.pho, satrec.pinco,
				satrec.plo,  rtemsq,        satrec.se2, satrec.se3,
				satrec.sgh2, satrec.sgh3,   satrec.sgh4,
				satrec.sh2,  satrec.sh3,    satrec.si2, satrec.si3,
				satrec.sl2,  satrec.sl3,    satrec.sl4, s1, s2, s3, s4, s5,
				s6,   s7,   ss1,  ss2,  ss3,  ss4,  ss5,  ss6,  ss7, sz1, sz2, sz3,
				sz11, sz12, sz13, sz21, sz22, sz23, sz31, sz32, sz33,
				satrec.xgh2, satrec.xgh3,   satrec.xgh4, satrec.xh2,
				satrec.xh3,  satrec.xi2,    satrec.xi3,  satrec.xl2,
				satrec.xl3,  satrec.xl4,    nm, z1, z2, z3, z11,
				z12, z13, z21, z22, z23, z31, z32, z33,
				satrec.zmol, satrec.zmos
				);
			dpper
				(
				satrec.e3, satrec.ee2, satrec.peo, satrec.pgho,
				satrec.pho, satrec.pinco, satrec.plo, satrec.se2,
				satrec.se3, satrec.sgh2, satrec.sgh3, satrec.sgh4,
				satrec.sh2, satrec.sh3, satrec.si2,  satrec.si3,
				satrec.sl2, satrec.sl3, satrec.sl4,  satrec.t,
				satrec.xgh2,satrec.xgh3,satrec.xgh4, satrec.xh2,
				satrec.xh3, satrec.xi2, satrec.xi3,  satrec.xl2,
				satrec.xl3, satrec.xl4, satrec.zmol, satrec.zmos, inclm, satrec.init,
				satrec.ecco, satrec.inclo, satrec.nodeo, satrec.argpo, satrec.mo
				);

			argpm  = 0.0;
			nodem  = 0.0;
			mm     = 0.0;

			dsinit
				(
				whichconst,
				cosim, emsq, satrec.argpo, s1, s2, s3, s4, s5, sinim, ss1, ss2, ss3, ss4,
				ss5, sz1, sz3, sz11, sz13, sz21, sz23, sz31, sz33, satrec.t, tc,
				satrec.gsto, satrec.mo, satrec.mdot, satrec.no, satrec.nodeo,
				satrec.nodedot, xpidot, z1, z3, z11, z13, z21, z23, z31, z33,
				satrec.ecco, eccsq, em, argpm, inclm, mm, nm, nodem,
				satrec.irez,  satrec.atime,
				satrec.d2201, satrec.d2211, satrec.d3210, satrec.d3222 ,
				satrec.d4410, satrec.d4422, satrec.d5220, satrec.d5232,
				satrec.d5421, satrec.d5433, satrec.dedt,  satrec.didt,
				satrec.dmdt,  dndt,         satrec.dnodt, satrec.domdt ,
				satrec.del1,  satrec.del2,  satrec.del3,  satrec.xfact,
				satrec.xlamo, satrec.xli,   satrec.xni
				);
		}

		/* ----------- set variables if not deep space ----------- */
		if (satrec.isimp != 1)
		{
			cc1sq          = satrec.cc1 * satrec.cc1;
			satrec.d2    = 4.0 * ao * tsi * cc1sq;
			temp           = satrec.d2 * tsi * satrec.cc1 / 3.0;
			satrec.d3    = (17.0 * ao + sfour) * temp;
			satrec.d4    = 0.5 * temp * ao * tsi * (221.0 * ao + 31.0 * sfour) *
				satrec.cc1;
			satrec.t3cof = satrec.d2 + 2.0 * cc1sq;
			satrec.t4cof = 0.25 * (3.0 * satrec.d3 + satrec.cc1 *
				(12.0 * satrec.d2 + 10.0 * cc1sq));
			satrec.t5cof = 0.2 * (3.0 * satrec.d4 +
				12.0 * satrec.cc1 * satrec.d3 +
				6.0 * satrec.d2 * satrec.d2 +
				15.0 * cc1sq * (2.0 * satrec.d2 + cc1sq));
		}
	} // if omeosq = 0 ...

	/* finally propogate to zero epoch to initialise all others. */
	if(satrec.error == 0)
		sgp4(whichconst, satrec, 0.0, r, v);

	satrec.init = 'n';

	//#include "debug6.cpp"
	return satrec.error;
}  // end sgp4init


///////////////////////////////////////////////////////////////////////////////
/// \brief add two vectors of size _count_
///
/// CUDA kernel
/// \param[in]  op1   term one
/// \param[in]  op2   term two
/// \param[in]  count vector size
/// \param[out] sum   result
///////////////////////////////////////////////////////////////////////////////
__global__ 
void AddKernel(const float *op1, const float *op2, int count, float *sum)
{
    const int pos = threadIdx.x + blockIdx.x * blockDim.x;

    if (pos >= count) return;

    sum[pos] = op1[pos] + op2[pos];
}

///////////////////////////////////////////////////////////////////////////////
/// \brief add two vectors of size _count_
/// \param[in]  op1   term one
/// \param[in]  op2   term two
/// \param[in]  count vector size
/// \param[out] sum   result
///////////////////////////////////////////////////////////////////////////////
static
void Add(const float *op1, const float *op2, int count, float *sum)
{
    dim3 threads(256);
    dim3 blocks(iDivUp(count, threads.x));

    AddKernel<<<blocks, threads>>>(op1, op2, count, sum);
}



static void initl
	(
	int satn,      gravconsttype whichconst,
	double ecco,   double epoch,  double inclo,   double& no,
	char& method,
	double& ainv,  double& ao,    double& con41,  double& con42, double& cosio,
	double& cosio2,double& eccsq, double& omeosq, double& posq,
	double& rp,    double& rteosq,double& sinio , double& gsto
	)
{
	/* --------------------- local variables ------------------------ */
	double ak, d1, del, adel, po, x2o3, j2, xke,
		tumin, mu, radiusearthkm, j3, j4, j3oj2;

	// sgp4fix use old way of finding gst
	int ids70;
	double ts70, ds70, tfrac, c1, thgr70, fk5r, c1p2p, thgr, thgro;
	const double twopi = 2.0 * pi;

	/* ----------------------- earth constants ---------------------- */
	// sgp4fix identify constants and allow alternate values
	getgravconst( whichconst, tumin, mu, radiusearthkm, xke, j2, j3, j4, j3oj2 );
	x2o3   = 2.0 / 3.0;

	/* ------------- calculate auxillary epoch quantities ---------- */
	eccsq  = ecco * ecco;
	omeosq = 1.0 - eccsq;
	rteosq = sqrt(omeosq);
	cosio  = cos(inclo);
	cosio2 = cosio * cosio;

	/* ------------------ un-kozai the mean motion ----------------- */
	ak    = pow(xke / no, x2o3);
	d1    = 0.75 * j2 * (3.0 * cosio2 - 1.0) / (rteosq * omeosq);
	del   = d1 / (ak * ak);
	adel  = ak * (1.0 - del * del - del *
		(1.0 / 3.0 + 134.0 * del * del / 81.0));
	del   = d1/(adel * adel);
	no    = no / (1.0 + del);

	ao    = pow(xke / no, x2o3);
	sinio = sin(inclo);
	po    = ao * omeosq;
	con42 = 1.0 - 5.0 * cosio2;
	con41 = -con42-cosio2-cosio2;
	ainv  = 1.0 / ao;
	posq  = po * po;
	rp    = ao * (1.0 - ecco);
	method = 'n';

	// sgp4fix modern approach to finding sidereal timew
	// gsto = gstime(epoch + 2433281.5);

	// sgp4fix use old way of finding gst
	// count integer number of days from 0 jan 1970
	ts70  = epoch - 7305.0;
	ids70 = floor(ts70 + 1.0e-8);
	ds70  = ids70;
	tfrac = ts70 - ds70;
	// find greenwich location at epoch
	c1    = 1.72027916940703639e-2;
	thgr70= 1.7321343856509374;
	fk5r  = 5.07551419432269442e-15;
	c1p2p = c1 + twopi;
	gsto  = fmod( thgr70 + c1*ds70 + c1p2p*tfrac + ts70*ts70*fk5r, twopi);
	if ( gsto < 0.0 )
		gsto = gsto + twopi;

	//#include "debug5.cpp"
}  // end initl